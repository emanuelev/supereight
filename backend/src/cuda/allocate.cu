#include "hip/hip_runtime.h"
#include "../common/field_impls.hpp"
#include "allocate.hpp"

#include <supereight/algorithms/unique.hpp>
#include <supereight/utils/cuda_util.hpp>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <chrono>

namespace se {

template<typename OctreeT>
__global__ static void buildAllocationListKernel(
    BufferAccessorCUDA<se::key_t> allocation_list, OctreeT octree,
    int* voxel_count, Eigen::Matrix4f pose, Eigen::Matrix4f K,
    BufferAccessorCUDA<float> depth, Eigen::Vector2i frame_size, float mu) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= frame_size.x() || y >= frame_size.y()) return;

    const Eigen::Matrix4f inv_P = pose * K.inverse();

    const float depth_sample = depth[x + y * frame_size.x()];
    if (depth_sample == 0) return;

    const Eigen::Vector3f camera_pos = pose.topRightCorner<3, 1>();
    Eigen::Vector3f world_vertex     = (inv_P *
        Eigen::Vector3f(
            (x + 0.5f) * depth_sample, (y + 0.5f) * depth_sample, depth_sample)
            .homogeneous())
                                       .head<3>();
    Eigen::Vector3f direction = (camera_pos - world_vertex).normalized();

    auto get_idx = [=]() { return atomicAdd(voxel_count, 1); };
    OctreeT::traits_type::buildAllocationList(allocation_list.data(),
        allocation_list.size(), get_idx, octree, world_vertex, direction,
        camera_pos, depth_sample, mu);
}

__global__ static void keysToLevelKernel(BufferAccessorCUDA<se::key_t> out,
    BufferAccessorCUDA<se::key_t> in, int num_elem, int level, int max_level) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elem) return;

    se::key_t key = in[idx];
    int key_level = keyops::level(key);

    unsigned shift = MAX_BITS - max_level - 1;
    se::key_t mask = MASK[level + shift];

    out[idx] = (key & mask & ~SCALE_MASK) | min(key_level, level);
}

template<typename OctreeT>
__global__ static void allocateLevelKernel(OctreeT octree,
    BufferAccessorCUDA<se::key_t> keys_at_level, int num_elem, int level) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elem) return;

    se::key_t key = keys_at_level[idx];
    int key_level = keyops::level(key);

    if (key_level < level) return;

    octree.insert_one(keys_at_level[idx], level);
}

template<typename OctreeT>
__global__ static void allocateSequentialKernel(OctreeT octree,
    BufferAccessorCUDA<se::key_t> allocation_list, int num_elem) {
    for (int i = 0; i < num_elem; ++i) {
        se::key_t key = allocation_list[i];

        int level           = keyops::level(key);
        Eigen::Vector3i pos = keyops::decode(key);

        octree.insert(pos.x(), pos.y(), pos.z(), level);
    }
}

int buildAllocationList(BufferAccessorCUDA<se::key_t> allocation_list,
    const Octree<FieldType, MemoryPoolCUDA>& octree, int* voxel_count,
    const Eigen::Matrix4f& pose, const Eigen::Matrix4f& K,
    BufferAccessorCUDA<float> depth, const Eigen::Vector2i& frame_size,
    float mu) {
    constexpr int thread_dim = 16;

    dim3 threads(thread_dim, thread_dim);
    dim3 blocks((frame_size.x() + thread_dim - 1) / thread_dim,
        (frame_size.y() + thread_dim - 1) / thread_dim);

    hipMemset(voxel_count, 0, sizeof(int));

    buildAllocationListKernel<<<blocks, threads>>>(
        allocation_list, octree, voxel_count, pose, K, depth, frame_size, mu);
    safeCall(hipPeekAtLastError());

    int final_count;
    hipMemcpy(&final_count, voxel_count, sizeof(int), hipMemcpyDeviceToHost);

    int reserved = allocation_list.size();
    return final_count >= reserved ? reserved : final_count;
}

int filterAllocationList(BufferAccessorCUDA<se::key_t> allocation_list,
    int allocation_list_used, int* num_unique_device,
    BufferCUDA<std::uint8_t>& temp_storage) {
    std::size_t temp_storage_bytes = 0;

    // Calculate temp storage requirements
    hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes,
        allocation_list.data(), allocation_list.data(), allocation_list_used);
    safeCall(hipPeekAtLastError());

    if (temp_storage_bytes > temp_storage.size())
        temp_storage.resize(temp_storage_bytes);

    // Sort
    hipcub::DeviceRadixSort::SortKeys(
        static_cast<void*>(temp_storage.accessor().data()), temp_storage_bytes,
        allocation_list.data(), allocation_list.data(), allocation_list_used);
    safeCall(hipPeekAtLastError());

    // Calculate temp storage requirements
    hipcub::DeviceSelect::Unique(nullptr, temp_storage_bytes,
        allocation_list.data(), allocation_list.data(), num_unique_device,
        allocation_list_used);
    safeCall(hipPeekAtLastError());

    if (temp_storage_bytes > temp_storage.size())
        temp_storage.resize(temp_storage_bytes);

    // Unique
    hipcub::DeviceSelect::Unique(temp_storage.accessor().data(),
        temp_storage_bytes, allocation_list.data(), allocation_list.data(),
        num_unique_device, allocation_list_used);
    safeCall(hipPeekAtLastError());

    int num_unique;
    safeCall(hipMemcpy(
        &num_unique, num_unique_device, sizeof(int), hipMemcpyDeviceToHost));

    return num_unique;
}

void allocateParallel(Octree<FieldType, MemoryPoolCUDA>& octree,
    BufferAccessorCUDA<se::key_t> allocation_list, int num_unique,
    BufferCUDA<se::key_t>& keys_at_level, int* keys_at_level_used,
    BufferCUDA<std::uint8_t>& temp_storage) {
    auto& node_buffer  = octree.getNodesBuffer();
    auto& block_buffer = octree.getBlockBuffer();

    std::size_t node_buffer_used  = node_buffer.used();
    std::size_t block_buffer_used = block_buffer.used();

    if (static_cast<std::size_t>(num_unique) > keys_at_level.size())
        keys_at_level.resize(num_unique);

    std::size_t temp_storage_bytes = 0;
    for (int level = 1; level <= octree.blockDepth(); ++level) {
        constexpr int thread_dim_ktl = 256;

        dim3 threads_ktl(thread_dim_ktl);
        dim3 blocks_ktl((num_unique + thread_dim_ktl - 1) / thread_dim_ktl);

        keysToLevelKernel<<<blocks_ktl, threads_ktl>>>(keys_at_level.accessor(),
            allocation_list, num_unique, level, octree.maxDepth());
        safeCall(hipPeekAtLastError());

        // Calculate temp storage requirements
        hipcub::DeviceSelect::Unique(nullptr, temp_storage_bytes,
            keys_at_level.accessor().data(), keys_at_level.accessor().data(),
            keys_at_level_used, num_unique);
        safeCall(hipPeekAtLastError());

        if (temp_storage_bytes > temp_storage.size())
            temp_storage.resize(temp_storage_bytes);

        // Unique keys_at_level
        hipcub::DeviceSelect::Unique(temp_storage.accessor().data(),
            temp_storage_bytes, keys_at_level.accessor().data(),
            keys_at_level.accessor().data(), keys_at_level_used, num_unique);
        safeCall(hipPeekAtLastError());

        int num_unique_at_level;
        safeCall(hipMemcpy(&num_unique_at_level, keys_at_level_used,
            sizeof(int), hipMemcpyDeviceToHost));
        if (num_unique_at_level == 0) continue;

        if (level < octree.blockDepth()) {
            node_buffer_used += num_unique_at_level;
            node_buffer.reserve(node_buffer_used);
        } else {
            block_buffer.reserve(block_buffer_used + num_unique_at_level);
        }

        constexpr int thread_dim_al = 256;

        dim3 threads_al(thread_dim_al);
        dim3 blocks_al(
            (num_unique_at_level + thread_dim_al - 1) / thread_dim_al);

        allocateLevelKernel<<<blocks_al, threads_al>>>(
            octree, keys_at_level.accessor(), num_unique_at_level, level);
        safeCall(hipPeekAtLastError());
    }
}

void allocate(Octree<FieldType, MemoryPoolCUDA>& octree,
    BufferAccessorCUDA<se::key_t> allocation_list, int allocation_list_used,
    BufferCUDA<se::key_t>& keys_at_level, int* keys_at_level_used,
    BufferCUDA<std::uint8_t>& temp_storage) {
    if (allocation_list_used == 0) return;

    // Parameters found using linear regression on GTX 1080 Ti test platform
    constexpr int filter_threshold   = 25;
    constexpr int parallel_threshold = 56;

    int num_elem;
    if (allocation_list_used < filter_threshold) {
        num_elem = allocation_list_used;
    } else {
        num_elem = filterAllocationList(allocation_list, allocation_list_used,
            keys_at_level_used, temp_storage);
    }

    if (num_elem < parallel_threshold) {
        auto& node_buffer  = octree.getNodesBuffer();
        auto& block_buffer = octree.getBlockBuffer();

        std::size_t node_buffer_used  = node_buffer.used();
        std::size_t block_buffer_used = block_buffer.used();

        node_buffer.reserve(node_buffer_used + (num_elem * octree.maxDepth()));
        block_buffer.reserve(block_buffer_used + num_elem);

        allocateSequentialKernel<<<1, 1>>>(octree, allocation_list, num_elem);
        safeCall(hipPeekAtLastError());
    } else {
        allocateParallel(octree, allocation_list, num_elem, keys_at_level,
            keys_at_level_used, temp_storage);
    }
}

} // namespace se
